#include "hip/hip_runtime.h"
#include<iostream>
#include<sys/time.h>
#include<stdlib.h>
#include<stdio.h>
#include<cassert>

#define ITERATIONS 10
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

using namespace std;

__global__ void spmv(float* row, float* col, float* data, float* vec, float* res, int dim, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<dim){
    float tmp = 0;
    
    #pragma unroll
    for(int j=row[i]; j<row[i+1]; j++){
      int colTmp = col[j];
      tmp +=  data[j] * vec[colTmp];
    }
    res[i] = tmp;
  }
}

double timestamp(){
  struct timeval tv;
  gettimeofday (&tv, 0);
  return tv.tv_sec + 1e-6*tv.tv_usec;
}
//The CSR-format matrix is dimXdim that has n non-zero elements.
void initMatrix(int *row, int *col, float *data, int n, int dim){
      int nnzAssigned = 0;

    // Figure out the probability that a nonzero should be assigned to a given
    // spot in the matrix
    double prob = (double)n / ((double)dim * (double)dim);

    // Seed random number generator
    srand48(8675309L);

    // Randomly decide whether entry i,j gets a value, but ensure n values
    // are assigned
     bool fillRemaining = false;
    for (int i = 0; i < dim; i++)
    {
        row[i] = nnzAssigned;
        for (int j = 0; j < dim; j++)
        {
            int numEntriesLeft = (dim * dim) - ((i * dim) + j);
            int needToAssign = n - nnzAssigned;
            if (numEntriesLeft <= needToAssign) {
                fillRemaining = true;
            }
            if ((nnzAssigned < n && drand48() <= prob) || fillRemaining)
            {
                // Assign (i,j) a value
                col[nnzAssigned] = j;
		data[nnzAssigned] = 1;
                nnzAssigned++;
            }
        }
   }
    // Observe the convention to put the number of non zeroes at the end of the
    // row delimiters array
    row[dim] = n;
    assert(nnzAssigned == n);
}



int main(){

  int dim=20000;
  int n=dim*dim/100;
  int *row = (int*)malloc(sizeof(int)*(dim+1));
  int *col = (int*)malloc(sizeof(int)*n);
  float *data = (float*)malloc(sizeof(float)*n);
  initMatrix(row, col, data, n, dim);

  float *vec = (float*)malloc(sizeof(float)*dim);

  for(int i=0; i<dim; i++){
    vec[i]=1;
  }

  float *result = (float*)malloc(sizeof(float)*dim);
  float *result_gpu_res = (float*)malloc(sizeof(float)*dim);

  for(int i=0; i<dim; i++){
    float t = 0;
    for(int j=row[i]; j<row[i+1]; j++){
      int colNum = col[j];
      t += data[j] * vec[colNum];
    }
    result[i] = t;
  }

  int *row_gpu;
  int *col_gpu;
  float *data_gpu;
  float *vec_gpu;
  float *result_gpu;
  hipMalloc( (void **)&row_gpu, sizeof(int)*(dim+1));
  hipMalloc( (void **)&col_gpu, sizeof(int)*n);
  hipMalloc( (void **)&data_gpu, sizeof(float)*n);
  hipMalloc( (void **)&vec_gpu, sizeof(float)*dim);
  hipMalloc( (void **)&result_gpu, sizeof(float)*dim);
  hipMemcpy(row_gpu, row, sizeof(int)*(dim+1), hipMemcpyHostToDevice);
  hipMemcpy(col_gpu, col, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(data_gpu, data, sizeof(float)*n, hipMemcpyHostToDevice);
  hipMemcpy(vec_gpu, vec, sizeof(float)*n, hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)ceil((float)dim/ ((float)DIM_THREAD_BLOCK_X)), 1);

  spmv<<<grid,block>>>(row_gpu, col_gpu, data_gpu, vec_gpu, result_gpu, dim, n);
  hipDeviceSynchronize();
  hipMemcpy(result_gpu_res, result_gpu, sizeof(float)*N*N, hipMemcpyDeviceToHost);
  compare(result, result_gpu_res, dim);



  double time1=timestamp();
  for(int numOfTimes=0; numOfTimes<ITERATIONS; numOfTimes++){

    spmv<<<grid,block>>>(row_gpu, col_gpu, data_gpu, vec_gpu, result_gpu, dim, n);

  }
  double time2=timestamp();

  double gflop = 2 * (double)n;

  double time = (time2-time1)/ITERATIONS;
  double flops = 2 * (double)n;
  double gflopsPerSecond = flops/(1000000000)/time;
  double dataCopy = sizeof(int)*dim + sizeof(int)*n + sizeof(float)*n + sizeof(float)*dim*2;
  double bandwidth = dataCopy/time/1000000000;
  printf("GFLOPS/s=%lf\n",gflopsPerSecond );
  printf("GB/s=%lf\n",bandwidth );
  printf("GB=%lf\n",dataCopy/1000000000);
  printf("GFLOPS=%lf\n",flops/(1000000000));
  printf("time(s)=%lf\n",time);
  return 0;
}
